#include "hip/hip_runtime.h"
#include "BuildGraph.h"

using namespace cv;
using namespace gpu;
using namespace device;

//template <typename T>
//__global__ void blendLinearKernel(int rows, int cols, int cn, const PtrStep<T> img1, const PtrStep<T> img2,
//								  const PtrStepf weights1, const PtrStepf weights2, PtrStep<T> result)
//{
//	int x = blockIdx.x * blockDim.x + threadIdx.x;
//	int y = blockIdx.y * blockDim.y + threadIdx.y;
//
//	if (y < rows && x < cols)
//	{
//		int x_ = x / cn;
//		float w1 = weights1.ptr(y)[x_];
//		float w2 = weights2.ptr(y)[x_];
//		T p1 = img1.ptr(y)[x];
//		T p2 = img2.ptr(y)[x];
//		result.ptr(y)[x] = (p1 * w1 + p2 * w2) / (w1 + w2 + 1e-5f);
//	}
//}
//
//template <typename T>
//void blendLinearCaller(int rows, int cols, int cn, PtrStep<T> img1, PtrStep<T> img2, PtrStepf weights1, PtrStepf weights2, PtrStep<T> result, hipStream_t stream)
//{
//	dim3 threads(16, 16);
//	dim3 grid(divUp(cols * cn, threads.x), divUp(rows, threads.y));
//
//	blendLinearKernel<<<grid, threads, 0, stream>>>(rows, cols * cn, cn, img1, img2, weights1, weights2, result);
//	cudaSafeCall( hipGetLastError() );
//
//	if (stream == 0)
//		cudaSafeCall(hipDeviceSynchronize());
//}
//
//template void blendLinearCaller<float>(int, int, int, PtrStep<float>, PtrStep<float>, PtrStepf, PtrStepf, PtrStep<float>, hipStream_t stream);

__global__ void igpuBuildGraphKernel(int safeWidth, int safeHeight, int width, PtrStep<float> R, PtrStep<float> G, PtrStep<float> B, PtrStep<float> D, Edge3D *edges) {
	int x = blockIdx.x * blockDim.x + threadIdx.x;
	int y = blockIdx.y * blockDim.y + threadIdx.y;
	int a = (y * width + x), iter = 4*a;
	if (x < safeWidth) {
		edges[iter].a = a;
		edges[iter].b = a + 1;
		edges[iter].w = fabsf(D.ptr(y)[x] - D.ptr(y)[x+1]);
		float rdiff = R.ptr(y)[x] - R.ptr(y)[x+1], gdiff = G.ptr(y)[x] - G.ptr(y)[x+1], bdiff = B.ptr(y)[x] - B.ptr(y)[x+1];
		edges[iter].w2 =  sqrtf(rdiff * rdiff + gdiff * gdiff + bdiff * bdiff);
		edges[iter].valid = true;
		iter++;
	}
	if (y < safeHeight) {
		edges[iter].a = a;
		edges[iter].b = a + width;
		edges[iter].w = fabsf(D.ptr(y)[x] - D.ptr(y+1)[x]);
		float rdiff = R.ptr(y)[x] - R.ptr(y+1)[x], gdiff = G.ptr(y)[x] - G.ptr(y+1)[x], bdiff = B.ptr(y)[x] - B.ptr(y+1)[x];
		edges[iter].w2 =  sqrtf(rdiff * rdiff + gdiff * gdiff + bdiff * bdiff);
		edges[iter].valid = true;
		iter++;
	}
	if (x < safeWidth && y < safeHeight) {
		edges[iter].a = a;
		edges[iter].b = a + width + 1;
		edges[iter].w = fabsf(D.ptr(y)[x] - D.ptr(y+1)[x+1]);
		float rdiff = R.ptr(y)[x] - R.ptr(y+1)[x+1], gdiff = G.ptr(y)[x] - G.ptr(y+1)[x+1], bdiff = B.ptr(y)[x] - B.ptr(y+1)[x+1];
		edges[iter].w2 =  sqrtf(rdiff * rdiff + gdiff * gdiff + bdiff * bdiff);
		edges[iter].valid = true;
		iter++;
	}
	if (x < safeWidth && y > 0) {
		edges[iter].a = a;
		edges[iter].b = a - width + 1;
		edges[iter].w = fabsf(D.ptr(y)[x] - D.ptr(y-1)[x+1]);
		float rdiff = R.ptr(y)[x] - R.ptr(y-1)[x+1], gdiff = G.ptr(y)[x] - G.ptr(y-1)[x+1], bdiff = B.ptr(y)[x] - B.ptr(y-1)[x+1];
		edges[iter].w2 =  sqrtf(rdiff * rdiff + gdiff * gdiff + bdiff * bdiff);
		edges[iter].valid = true;
	}
}

void igpuBuildGraph(Mat &R, Mat &G, Mat &B, Mat &D, Edge3D *edges, int numEdges) {
	//hipSetDevice(0);
	dim3 threads(16,16);
	int cols = R.cols, rows = R.rows;
	dim3 grid(divUp(cols, threads.x), divUp(rows, threads.y));

	int widthMinus = cols - 1, heightMinus = rows - 1;
	GpuMat gpuR(R), gpuG(G), gpuB(B), gpuD(D);
	Edge3D *gpuEdges;
	size_t edge_size = numEdges*sizeof(Edge3D);
	hipMalloc(&gpuEdges, edge_size);
	hipMemset(gpuEdges,0,edge_size);
	hipMemcpy(gpuEdges,edges,edge_size,hipMemcpyHostToDevice);
	igpuBuildGraphKernel<<<grid, threads>>>(widthMinus, heightMinus, cols, gpuR, gpuG, gpuB, gpuD, gpuEdges);
	cudaSafeCall( hipGetLastError() );
	cudaSafeCall(hipDeviceSynchronize());
	hipMemcpy(edges,gpuEdges,edge_size,hipMemcpyDeviceToHost);
	hipFree(gpuEdges);
}

void thrustsort(Edge3D *pEdge, Edge3D *edgesEnd) {
	thrust::device_vector<Edge3D> d_vec;
	thrust::copy(pEdge,edgesEnd,d_vec.begin());
	thrust::sort(d_vec.begin(),d_vec.end());
	thrust::copy(d_vec.begin(),d_vec.end(),pEdge);
}

__device__ bool lessThan3DGPU(const Edge3D& a, const Edge3D& b) {
	return a.w2 < b.w2;
}

void thrustsort2(Edge3D *pEdge, Edge3D *edgesEnd) {
	thrust::device_vector<Edge3D> d_vec;
	thrust::copy(pEdge,edgesEnd,d_vec.begin());
	thrust::sort(d_vec.begin(),d_vec.end(), lessThan3DGPU);
	thrust::copy(d_vec.begin(),d_vec.end(),pEdge);
}